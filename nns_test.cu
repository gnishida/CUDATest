﻿/**
 * Nearest neighbor search
 * マップ内に店ゾーンが20%の確率で配備されている時、
 * 住宅ゾーンから直近の店ゾーンまでのマンハッタン距離を計算する。
 * Kd-treeなどのアルゴリズムだと、各住宅ゾーンから直近の店までの距離の計算にO(log M)。
 * 従って、全ての住宅ゾーンについて調べると、O(N log M)。
 * 一方、本実装では、各店ゾーンから周辺ゾーンに再帰的に距離を更新していくので、O(N)で済む。
 * しかも、GPUで並列化することで、さらに計算時間を短縮できる。
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <list>
#include <time.h>

#define CITY_SIZE 400
#define NUM_GPU_BLOCKS 4
#define NUM_GPU_THREADS 32
#define NUM_FEATURES 1



struct ZoneType {
	int type;
	int level;
};

struct ZoningPlan {
	ZoneType zones[CITY_SIZE][CITY_SIZE];
};

struct DistanceMap {
	int distances[CITY_SIZE][CITY_SIZE][NUM_FEATURES];
};

struct Point2D {
	int x;
	int y;

	__host__ __device__
	Point2D() : x(0), y(0) {}

	__host__ __device__
	Point2D(int x, int y) : x(x), y(y) {}
};


__host__ __device__
unsigned int rand(unsigned int* randx) {
    *randx = *randx * 1103515245 + 12345;
    return (*randx)&2147483647;
}

__host__ __device__
float randf(unsigned int* randx) {
	return rand(randx) / (float(2147483647) + 1);
}

__host__ __device__
float randf(unsigned int* randx, float a, float b) {
	return randf(randx) * (b - a) + a;
}

__host__ __device__
int sampleFromCdf(unsigned int* randx, float* cdf, int num) {
	float rnd = randf(randx, 0, cdf[num-1]);

	for (int i = 0; i < num; ++i) {
		if (rnd <= cdf[i]) return i;
	}

	return num - 1;
}

__host__ __device__
int sampleFromPdf(unsigned int* randx, float* pdf, int num) {
	if (num == 0) return 0;

	float cdf[40];
	cdf[0] = pdf[0];
	for (int i = 1; i < num; ++i) {
		if (pdf[i] >= 0) {
			cdf[i] = cdf[i - 1] + pdf[i];
		} else {
			cdf[i] = cdf[i - 1];
		}
	}

	return sampleFromCdf(randx, cdf, num);
}

/**
 * ゾーンプランを生成する。
 */
__host__
void generateZoningPlan(ZoningPlan& zoningPlan, std::vector<float> zoneTypeDistribution, std::vector<Point2D>& hostStoreLocations) {
	std::vector<float> numRemainings(zoneTypeDistribution.size());
	for (int i = 0; i < zoneTypeDistribution.size(); ++i) {
		numRemainings[i] = CITY_SIZE * CITY_SIZE * zoneTypeDistribution[i];
	}

	unsigned int randx = 0;

	for (int r = 0; r < CITY_SIZE; ++r) {
		for (int c = 0; c < CITY_SIZE; ++c) {
			int type = sampleFromPdf(&randx, numRemainings.data(), numRemainings.size());
			zoningPlan.zones[r][c].type = type;
			numRemainings[type] -= 1;

			switch (type) {
			case 0:
				break;
			case 1:
				hostStoreLocations.push_back(Point2D(c, r));
				break;
			}
		}
	}
}



/**
 * 直近の店までの距離を計算する
 */
__global__
void computeDistanceToStore(ZoningPlan* zoningPLan, Point2D* storeLocations, int* numStores, DistanceMap* distanceMap) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	Point2D queue[1000];
	int queue_begin = 0;
	int queue_end = 0;

	int stride = ceilf((float)*numStores / NUM_GPU_BLOCKS / NUM_GPU_THREADS);

	for (int i = 0; i < stride; ++i) {
		int store_id = idx * stride + i;
		if (store_id >= *numStores) break;

		queue[queue_end++] = storeLocations[store_id];
		distanceMap->distances[storeLocations[store_id].y][storeLocations[store_id].x][0] = 0;
	}

	while (queue_begin < queue_end) {
		Point2D pt = queue[queue_begin++];

		int d = distanceMap->distances[pt.y][pt.x][0];

		if (pt.y > 0) {
			int old = atomicMin(&distanceMap->distances[pt.y-1][pt.x][0], d + 1);
			if (old > d + 1) {
				queue[queue_end++] = Point2D(pt.x, pt.y-1);
			}
		}
		if (pt.y < CITY_SIZE - 1) {
			int old = atomicMin(&distanceMap->distances[pt.y+1][pt.x][0], d + 1);
			if (old > d + 1) {
				queue[queue_end++] = Point2D(pt.x, pt.y+1);
			}
		}
		if (pt.x > 0) {
			int old = atomicMin(&distanceMap->distances[pt.y][pt.x-1][0], d + 1);
			if (old > d + 1) {
				queue[queue_end++] = Point2D(pt.x-1, pt.y);
			}
		}
		if (pt.x < CITY_SIZE - 1) {
			int old = atomicMin(&distanceMap->distances[pt.y][pt.x+1][0], d + 1);
			if (old > d + 1) {
				queue[queue_end++] = Point2D(pt.x+1, pt.y);
			}
		}
	}
}

/**
 * 直近の店までの距離を計算する
 */
__global__
void computeDistanceToStoreBySingleThread(ZoningPlan* zoningPLan, Point2D* storeLocations, int* numStores, DistanceMap* distanceMap) {
	Point2D queue[1000];
	int queue_begin = 0;
	int queue_end = 0;

	for (int i = 0; i < *numStores; ++i) {
		queue[queue_end++] = storeLocations[i];
		distanceMap->distances[storeLocations[i].y][storeLocations[i].x][0] = 0;
	}

	while (queue_begin < queue_end) {
		Point2D pt = queue[queue_begin++];

		int d = distanceMap->distances[pt.y][pt.x][0];

		if (pt.y > 0) {
			int old = atomicMin(&distanceMap->distances[pt.y-1][pt.x][0], d + 1);
			if (old > d + 1) {
				queue[queue_end++] = Point2D(pt.x, pt.y-1);
			}
		}
		if (pt.y < CITY_SIZE - 1) {
			int old = atomicMin(&distanceMap->distances[pt.y+1][pt.x][0], d + 1);
			if (old > d + 1) {
				queue[queue_end++] = Point2D(pt.x, pt.y+1);
			}
		}
		if (pt.x > 0) {
			int old = atomicMin(&distanceMap->distances[pt.y][pt.x-1][0], d + 1);
			if (old > d + 1) {
				queue[queue_end++] = Point2D(pt.x-1, pt.y);
			}
		}
		if (pt.x < CITY_SIZE - 1) {
			int old = atomicMin(&distanceMap->distances[pt.y][pt.x+1][0], d + 1);
			if (old > d + 1) {
				queue[queue_end++] = Point2D(pt.x+1, pt.y);
			}
		}
	}
}

int main()
{
	time_t start, end;


	ZoningPlan* hostZoningPlan = (ZoningPlan*)malloc(sizeof(ZoningPlan));
	std::vector<Point2D> hostStoreLocations;
	DistanceMap* hostDistanceMap = (DistanceMap*)malloc(sizeof(DistanceMap));
	DistanceMap* hostDistanceMap2 = (DistanceMap*)malloc(sizeof(DistanceMap));

	// 距離を初期化
	memset(hostDistanceMap, 9999, sizeof(DistanceMap));
	memset(hostDistanceMap2, 9999, sizeof(DistanceMap));

	std::vector<float> zoneTypeDistribution(2);
	zoneTypeDistribution[0] = 0.8f;
	zoneTypeDistribution[1] = 0.2f;
	
	// 初期プランを生成
	// 同時に、店の座標リストを作成
	start = clock();
	generateZoningPlan(*hostZoningPlan, zoneTypeDistribution, hostStoreLocations);
	end = clock();
	printf("generateZoningPlan: %lf\n", (double)(end-start)/CLOCKS_PER_SEC);
	/*
	for (int r = CITY_SIZE - 1; r >= 0; --r) {
		for (int c = 0; c < CITY_SIZE; ++c) {
			printf("%d, ", hostZoningPlan.zones[r][c].type);
		}
		printf("\n");
	}
	printf("\n");
	*/

	// 初期プランをデバイスバッファへコピー
	ZoningPlan* devZoningPlan;
	if (hipMalloc((void**)&devZoningPlan, sizeof(ZoningPlan)) != hipSuccess) {
		printf("memory allocation error!\n");
		exit(1);
	}
	if (hipMemcpy(devZoningPlan, hostZoningPlan, sizeof(ZoningPlan), hipMemcpyHostToDevice) != hipSuccess) {
		printf("memory copy error!\n");
		exit(1);
	}

	// 店の数をデバイスバッファへコピー
	int* devNumStores;
	if (hipMalloc((void**)&devNumStores, sizeof(int)) != hipSuccess) {
		printf("memory allocation error!\n");
		exit(1);
	}
	int hostNumStores = hostStoreLocations.size();
	if (hipMemcpy(devNumStores, &hostNumStores, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
		printf("memory copy error!\n");
		exit(1);
	}

	// 店の座標をデバイスバッファへコピー
	Point2D* devStoreLocations;
	if (hipMalloc((void**)&devStoreLocations, sizeof(Point2D) * hostNumStores) != hipSuccess) {
		printf("memory allocation error!\n");
		exit(1);
	}
	if (hipMemcpy(devStoreLocations, hostStoreLocations.data(), sizeof(Point2D) * hostNumStores, hipMemcpyHostToDevice) != hipSuccess) {
		printf("memory copy error!\n");
		exit(1);
	}


	// 距離マップ用に、デバイスバッファを確保
	DistanceMap* devDistanceMap;
	hipMalloc((void**)&devDistanceMap, sizeof(DistanceMap));


	///////////////////////////////////////////////////////////////////////
	// シングルスレッドで、直近の店までの距離を計算

	// 距離をデバイスバッファへコピー
	hipMemcpy(devDistanceMap, hostDistanceMap2, sizeof(DistanceMap), hipMemcpyHostToDevice);

	// スコアの直近の店までの距離を計算
	start = clock();
	computeDistanceToStoreBySingleThread<<<1, 1>>>(devZoningPlan, devStoreLocations, devNumStores, devDistanceMap);
	end = clock();
	printf("computeDistanceToStoreBySingleThread: %lf\n", (double)(end-start)/CLOCKS_PER_SEC);

	// 距離をCPUバッファへコピー
	hipMemcpy(hostDistanceMap2, devDistanceMap, sizeof(DistanceMap), hipMemcpyDeviceToHost);

	///////////////////////////////////////////////////////////////////////
	// マルチスレッドで、直近の店までの距離を計算

	// 距離をデバイスバッファへコピー
	hipMemcpy(devDistanceMap, hostDistanceMap, sizeof(DistanceMap), hipMemcpyHostToDevice);

	// スコアの直近の店までの距離を並列で計算
	start = clock();
	computeDistanceToStore<<<NUM_GPU_BLOCKS, NUM_GPU_THREADS>>>(devZoningPlan, devStoreLocations, devNumStores, devDistanceMap);
	end = clock();
	printf("computeDistanceToStore: %lf\n", (double)(end-start)/CLOCKS_PER_SEC);

	// 距離をCPUバッファへコピー
	hipMemcpy(hostDistanceMap, devDistanceMap, sizeof(DistanceMap), hipMemcpyDeviceToHost);


	
	// シングルスレッドとマルチスレッドの結果を比較
	for (int r = CITY_SIZE - 1; r >= 0; --r) {
		for (int c = 0; c < CITY_SIZE; ++c) {
			if (hostDistanceMap->distances[r][c][0] != hostDistanceMap2->distances[r][c][0]) {
				printf("ERROR!\n");
			}
		}
	}
	printf("\n");


	// デバイスバッファの開放
	hipFree(devZoningPlan);
	hipFree(devNumStores);
	hipFree(devStoreLocations);
	hipFree(devDistanceMap);

	// CPUバッファの開放
	free(hostZoningPlan);
	free(hostDistanceMap);
	free(hostDistanceMap2);

	hipDeviceReset();
}
