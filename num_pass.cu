﻿ 
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void add(int a, int b, int *c) {
    *c = a+ b;
}

int main(void) {
    int c;
    int *dev_c;

	// GPU側でint型の値を１個格納するためのメモリを確保する
    hipMalloc((void**)&dev_c, sizeof(int));

	// GPU側の関数を呼び出す。（）内が、そのまま関数の引数となる
    add<<<1,1>>>(12, 2000, dev_c);

	// 指定したsize分、GPUのdev_cから、CPUのcへ、データ（int型の値１個）を転送する
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	// GPU側で確保したメモリを開放する
    hipFree(dev_c);

	// 結果を出力する
    printf("12 + 2000 = %d\n", c);

	hipDeviceReset();

    return 0;
}