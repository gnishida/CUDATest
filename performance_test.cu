#include "hip/hip_runtime.h"
﻿#include <vector>
#include <iostream>

#define GRID_SIZE	200

struct zone_type {
	int type;
	int level;
};

struct zone_plan {
	zone_type zones[GRID_SIZE][GRID_SIZE];
	float score;
};

__host__ __device__
void MCMC(int numIterations) {
	float count = 0.0;

	zone_plan* hoge = (zone_plan*)malloc(sizeof(zone_plan));
	zone_plan* hoge2 = (zone_plan*)malloc(sizeof(zone_plan));
	for (int i = 0; i < numIterations; ++i) {
		memcpy(hoge, hoge2, sizeof(zone_plan));
		for (int r = 0; r < GRID_SIZE; ++r) {
			for (int c = 0; c < GRID_SIZE; ++c) {
			}
		}
	}
}

/**
 * CUDA version of MCMCM
 */
__global__
void MCMCGPUKernel(int* numIterations) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	// initialize random
	unsigned int randx = idx;

	MCMC(*numIterations);
}

/**
 * CUDA version of MCMC
 */
__host__
void zonePlanMCMCGPUfunc(int numIterations) {
	int* devNumIterations;
	if (hipMalloc((void**)&devNumIterations, sizeof(int)) != hipSuccess) {
		printf("cuda memory allocation error!\n");
		return;
	}

	if (hipMemcpy(devNumIterations, &numIterations, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
		printf("cuda memory copy error!\n");
		return;
	}

	// start kernel
	time_t start = clock();
    MCMCGPUKernel<<<1, 1>>>(devNumIterations);
	hipDeviceSynchronize();
	time_t end = clock();
	printf("Time elapsed: %lf\n", (double)(end-start)/CLOCKS_PER_SEC);
}

void main() {
	zonePlanMCMCGPUfunc(10000);

	// CPU
	time_t start = clock();
	MCMC(10000);
	time_t end = clock();
	printf("CPU version took %lf\n", (double)(end-start)/CLOCKS_PER_SEC);
}