#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <vector>

#define CITY_SIZE 200 //200
#define BLOCK_SIZE 40 //200
#define NUM_THREADS 32
#define MAX_ITERATIONS 1000
#define QUEUE_MAX 39999

__global__
void testShared(int* zone, int* dist) {
	__shared__ int sub_zone[BLOCK_SIZE * BLOCK_SIZE];
	__shared__ int sub_dist[BLOCK_SIZE * BLOCK_SIZE];

	// 
	int x0 = blockIdx.x * BLOCK_SIZE;
	int y0 = blockIdx.y * BLOCK_SIZE;

	for (int i = threadIdx.x; i < BLOCK_SIZE * BLOCK_SIZE; i += NUM_THREADS) {
		int x = i % BLOCK_SIZE;
		int y = i / BLOCK_SIZE;

		sub_zone[x + y * CITY_SIZE] = zone[x + x0 + (y + y0) * CITY_SIZE];
	}

	__syncthreads();


	for (int i = threadIdx.x; i < BLOCK_SIZE * BLOCK_SIZE; i += NUM_THREADS) {
		int x = i % BLOCK_SIZE;
		int y = i / BLOCK_SIZE;

		int total = sub_zone[x + y * BLOCK_SIZE];
		int count = 1;
		if (y > 0) {
			total += sub_zone[x + (y - 1) * BLOCK_SIZE];
			count++;
		}
		if (y < BLOCK_SIZE - 1) {
			total += sub_zone[x + (y + 1) * BLOCK_SIZE];
			count++;
		}
		if (x > 0) {
			total += sub_zone[x - 1 + y * BLOCK_SIZE];
			count++;
		}
		if (x < BLOCK_SIZE - 1) {
			total += sub_zone[x + 1 + y * BLOCK_SIZE];
			count++;
		}

		sub_dist[x + y * CITY_SIZE] = total / count;
	}


	__syncthreads();

	for (int i = threadIdx.x; i < BLOCK_SIZE * BLOCK_SIZE; i += NUM_THREADS) {
		int x = i % BLOCK_SIZE;
		int y = i / BLOCK_SIZE;

		dist[x + x0 + (y + y0) * CITY_SIZE] = sub_dist[x + y * CITY_SIZE]; 
	}
}

__global__
void testGlobal(int* zone, int* dist) {
	// 
	int x0 = blockIdx.x * BLOCK_SIZE;
	int y0 = blockIdx.y * BLOCK_SIZE;

	for (int i = threadIdx.x; i < BLOCK_SIZE * BLOCK_SIZE; i += NUM_THREADS) {
		int x = i % BLOCK_SIZE + x0;
		int y = i / BLOCK_SIZE + y0;

		int total = zone[x + y * CITY_SIZE];
		int count = 1;
		if (y > 0) {
			total += zone[x + (y - 1) * CITY_SIZE];
			count++;
		}
		if (y < CITY_SIZE - 1) {
			total += zone[x + (y + 1) * CITY_SIZE];
			count++;
		}
		if (x > 0) {
			total += zone[x - 1 + y * CITY_SIZE];
			count++;
		}
		if (x < CITY_SIZE - 1) {
			total += zone[x + 1 + y * CITY_SIZE];
			count++;
		}

		dist[x + y * CITY_SIZE] = total / count;
	}
}

__host__
void testCPU(int* zone, int* dist) {
	for (int r = 0; r < CITY_SIZE; ++r) {
		for (int c = 0; c < CITY_SIZE; ++c) {
			int total = zone[r * CITY_SIZE + c];
			int count = 1;
			if (r > 0) {
				total += zone[(r - 1) * CITY_SIZE + c];
				count++;
			}
			if (r < CITY_SIZE - 1) {
				total += zone[(r + 1) * CITY_SIZE + c];
				count++;
			}
			if (c > 0) {
				total += zone[r * CITY_SIZE + c - 1];
				count++;
			}
			if (c > CITY_SIZE - 1) {
				total += zone[r * CITY_SIZE + c + 1];
				count++;
			}

			dist[r * CITY_SIZE + c] = total / count;
		}
	}
}

__global__
void testQueueGlobal(int* zone, int* dist, int* queue, unsigned int* q_head, unsigned int* q_tail) {
	int x0 = blockIdx.x * BLOCK_SIZE;
	int y0 = blockIdx.y * BLOCK_SIZE;

	int stride = BLOCK_SIZE * BLOCK_SIZE / NUM_THREADS;
	int x = threadIdx.x * stride % BLOCK_SIZE;
	int y = threadIdx.x * stride / BLOCK_SIZE;

	unsigned int q_index = atomicInc(q_tail, QUEUE_MAX);
	queue[q_index] = x + x0 + (y + y0) * CITY_SIZE;
	dist[x + x0 + (y + y0) * CITY_SIZE] = 0;

	__syncthreads();

	while (true) {
		int q_index = atomicInc(q_head, QUEUE_MAX);
		int s = queue[q_index];
		if (s == -1) break;

		int x = s % CITY_SIZE;
		int y = s / CITY_SIZE;
		int d = dist[s];

		if (x > 0) {
			int old = atomicMin(&dist[s - 1], d + 1);
			if (old > d + 1) {
				unsigned int q_index2 = atomicInc(q_tail, QUEUE_MAX);
				queue[q_index2] = s - 1;
			}
		}
		if (x < CITY_SIZE - 1) {
			int old = atomicMin(&dist[s + 1], d + 1);
			if (old > d + 1) {
				unsigned int q_index2 = atomicInc(q_tail, QUEUE_MAX);
				queue[q_index2] = s + 1;
			}
		}
		if (y > 0) {
			int old = atomicMin(&dist[s - CITY_SIZE], d + 1);
			if (old > d + 1) {
				unsigned int q_index2 = atomicInc(q_tail, QUEUE_MAX);
				queue[q_index2] = s - CITY_SIZE;
			}
		}
		if (y < CITY_SIZE - 1) {
			int old = atomicMin(&dist[s + CITY_SIZE], d + 1);
			if (old > d + 1) {
				unsigned int q_index2 = atomicInc(q_tail, QUEUE_MAX);
				queue[q_index2] = s + CITY_SIZE;
			}
		}

		queue[q_index] = -1;
	}
}

__host__
void testQueueCPU(int* zone, int* dist) {
	std::vector<int> queue(QUEUE_MAX + 1);
	unsigned int q_head = 0;
	unsigned int q_tail = 0;

	for (int i = 0; i < QUEUE_MAX + 1; ++i) queue[i] = -1;

	for (int i = 0; i < 10; ++i) {
		int stride = CITY_SIZE * CITY_SIZE / 10;
		int x = i * stride % CITY_SIZE;
		int y = i * stride / CITY_SIZE;
		
		queue[q_tail++] = x + y * CITY_SIZE;
		dist[x + y * CITY_SIZE] = 0;
	}

	while (true) {
		int s = queue[q_head];
		if (s == -1) break;

		int x = s % CITY_SIZE;
		int y = s / CITY_SIZE;
		int d = dist[s];

		if (x > 0) {
			if (dist[s - 1] > d + 1) {
				dist[s - 1] = d + 1;
				queue[q_tail++] = s - 1;
			}
		}
		if (x < CITY_SIZE - 1) {
			if (dist[s + 1] > d + 1) {
				dist[s + 1] = d + 1;
				queue[q_tail++] = s + 1;
			}
		}
		if (y > 0) {
			if (dist[s - CITY_SIZE] > d + 1) {
				dist[s - CITY_SIZE] = d + 1;
				queue[q_tail++] = s - CITY_SIZE;
			}
		}
		if (y < CITY_SIZE - 1) {
			if (dist[s + CITY_SIZE] > d + 1) {
				dist[s + CITY_SIZE] = d + 1;
				queue[q_tail++] = s + CITY_SIZE;
			}
		}

		queue[q_head++] = -1;
	}
}

__host__
void showZone(int* devZone) {
	int* zone;
	zone = (int*)malloc(sizeof(int) * CITY_SIZE * CITY_SIZE);
	hipMemcpy(zone, devZone, sizeof(int) * CITY_SIZE * CITY_SIZE, hipMemcpyDeviceToHost);

	printf("<<< Zone Map >>>\n");
	for (int r = 0; r < CITY_SIZE; ++r) {
		for (int c = 0; c < CITY_SIZE; ++c) {
			printf("%2d,", zone[r * CITY_SIZE + c]);
		}
		printf("\n");
	}
	printf("\n");

	free(zone);
}

__host__
void showDist(int* devDist) {
	int* dist;
	dist = (int*)malloc(sizeof(int) * CITY_SIZE * CITY_SIZE);
	hipMemcpy(dist, devDist, sizeof(int) * CITY_SIZE * CITY_SIZE, hipMemcpyDeviceToHost);

	printf("<<< Distance Map >>>\n");
	for (int r = 0; r < CITY_SIZE; ++r) {
		for (int c = 0; c < CITY_SIZE; ++c) {
			printf("%2d,", dist[r * CITY_SIZE + c]);
		}
		printf("\n");
	}
	printf("\n");

	free(dist);
}

int main(int argc, char **argv) {
	time_t start, end;

	int* hostZone;
	hostZone = (int*)malloc(sizeof(int) * CITY_SIZE * CITY_SIZE);
	for (int i = 0; i < CITY_SIZE * CITY_SIZE; ++i) {
		hostZone[i] = rand() % 6;
	}
	int* hostDist;
	hostDist = (int*)malloc(sizeof(int) * CITY_SIZE * CITY_SIZE);

	int* devZone;
	hipMalloc((void**)&devZone, sizeof(int) * CITY_SIZE * CITY_SIZE);
	int* devDist;
	hipMalloc((void**)&devDist, sizeof(int) * CITY_SIZE * CITY_SIZE);


	hipMemcpy(devZone, hostZone, sizeof(int) * CITY_SIZE * CITY_SIZE, hipMemcpyHostToDevice);

	start = clock();
	for (int iter = 0; iter < MAX_ITERATIONS; ++iter) {
		testGlobal<<<dim3(CITY_SIZE / BLOCK_SIZE, CITY_SIZE / BLOCK_SIZE), NUM_THREADS>>>(devZone, devDist);
		hipDeviceSynchronize();
	}
	end = clock();
	printf("GPU (global): %lf\n", (double)(end-start)/CLOCKS_PER_SEC);

	start = clock();
	for (int iter = 0; iter < MAX_ITERATIONS; ++iter) {
		testShared<<<dim3(CITY_SIZE / BLOCK_SIZE, CITY_SIZE / BLOCK_SIZE), NUM_THREADS>>>(devZone, devDist);
		hipDeviceSynchronize();
	}
	end = clock();
	printf("GPU (shared): %lf\n", (double)(end-start)/CLOCKS_PER_SEC);

	start = clock();
	for (int iter = 0; iter < MAX_ITERATIONS; ++iter) {
		testCPU(hostZone, hostDist);
	}
	end = clock();
	printf("CPU: %lf\n", (double)(end-start)/CLOCKS_PER_SEC);
	//showZone(devZone);
	//showDist(devDist);



	int* devQueue;
	hipMalloc((void**)&devQueue, sizeof(int) * (QUEUE_MAX + 1));
	unsigned int* devQueueHead;
	hipMalloc((void**)&devQueueHead, sizeof(unsigned int));
	unsigned int* devQueueTail;
	hipMalloc((void**)&devQueueTail, sizeof(unsigned int));
	
	start = clock();
	for (int iter = 0; iter < MAX_ITERATIONS; ++iter) {
		hipMemset(devQueue, -1, sizeof(int) * (QUEUE_MAX + 1));
		hipMemset(devDist, 99, sizeof(int) * CITY_SIZE * CITY_SIZE);
		hipMemset(devQueueHead, 0, sizeof(unsigned int));
		hipMemset(devQueueTail, 0, sizeof(unsigned int));
		testQueueGlobal<<<dim3(CITY_SIZE / BLOCK_SIZE, CITY_SIZE / BLOCK_SIZE), NUM_THREADS>>>(devZone, devDist, devQueue, devQueueHead, devQueueTail);
		hipDeviceSynchronize();
	}
	end = clock();
	printf("GPU (global): %lf\n", (double)(end-start)/CLOCKS_PER_SEC);

	start = clock();
	for (int iter = 0; iter < MAX_ITERATIONS; ++iter) {
		hipMemset(devQueue, -1, sizeof(int) * (QUEUE_MAX + 1));
		hipMemset(devDist, 99, sizeof(int) * CITY_SIZE * CITY_SIZE);
		hipMemset(devQueueTail, 0, sizeof(unsigned int));
		testQueueCPU(hostZone, hostDist);
		hipDeviceSynchronize();
	}
	end = clock();
	printf("GPU (global): %lf\n", (double)(end-start)/CLOCKS_PER_SEC);
}